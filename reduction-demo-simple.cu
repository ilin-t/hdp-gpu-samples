
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <random>

#define N 8

// Error checking for CUDA methods (not for the kernel though)
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void SimpleSumReductionKernel (int* input, int* output){
    unsigned int i = 2*threadIdx.x;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2){
        if (threadIdx.x % stride == 0){
            input[i] += input[i + stride]; 
        }
        __syncthreads();
    }
    

    
    if(threadIdx.x == 0){
    	//for(int i=0; i<blockDim.x; i++){
    		//printf("%d \t", input[i]);
    	//}
        *output = input[0];
        printf("Output: %d \n", *output);
    }
}

int main(){


	std::cout<<"Size: "<< N*sizeof(int)<<" B\n";
	
	// start general timer 
	const auto start = std::chrono::steady_clock::now();
	
	int hx[N] = {1,8,3,2,5,7,4,6};
	int *hy = 0;
	
	int *gx, *gy;
	
	cudaErrorCheck( hipMalloc((void**)&gx, N*sizeof(int)));
	cudaErrorCheck( hipMalloc((void**)&gy, sizeof(int)));
	
	 // end for malloc
	const auto mem_alloc_end = std::chrono::steady_clock::now();
	
	// copy the arrays from the CPU to the GPU
	hipMemcpy(gx, hx, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(hx, gx, sizeof(int), hipMemcpyDeviceToHost);

	// end for initialization and memcpy
	const auto data_init_end = std::chrono::steady_clock::now();

	// Run kernel on 1 GiB of data on a single thread on the GPU
	SimpleSumReductionKernel<<<1, 4>>>(gx, gy);

	cudaErrorCheck( hipPeekAtLastError() ); //debug the kernel output
	cudaErrorCheck( hipDeviceSynchronize() );

	const auto kernel_end = std::chrono::steady_clock::now();

	// copy the resulting array back to the GPU
	hipMemcpy(&hy, &gy, sizeof(int), hipMemcpyDeviceToHost);

	// Free memory
	hipFree(gx);
	hipFree(gy);

	

	//delete [] hx;
	delete [] hy;

	const auto free_mem = std::chrono::steady_clock::now();
	

	std::cout << "Memory allocation duration: " << std::chrono::duration_cast<std::chrono::microseconds>(mem_alloc_end-start).count()/1000<< " ms \n" \
		    << "Data initialization duration: " <<  std::chrono::duration_cast<std::chrono::microseconds>(data_init_end-mem_alloc_end).count()/1000 << " ms \n" \
		    << "Kernel duration: " << std::chrono::duration_cast<std::chrono::microseconds>(kernel_end-data_init_end).count() << " microseconds \n" \
		    << "Free memory duration: " << std::chrono::duration_cast<std::chrono::microseconds>(free_mem - kernel_end).count()/1000 << " ms \n";
		    
	hipDeviceReset();

	return 0;
}
