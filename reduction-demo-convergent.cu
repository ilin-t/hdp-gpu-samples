
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <random>

#define N 8

// Error checking for CUDA methods (not for the kernel though)
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void ConvergentSumReductionKernel(int* gx, int* gy){
	unsigned int i = threadIdx.x;
	for (unsigned int stride = blockDim.x; stride >=1; stride /= 2){
		if (threadIdx.x < stride){
			gx[i] += gx[i + stride];
		}
		__syncthreads();
	}
	if(threadIdx.x == 0){
		*gy = gx[0];
	}
}

int main(){


	//int N = 1<<29;
	//std::cout<<"Size: "<< N*sizeof(int)<<" B\n";
	

	std::cout<<"Size: "<< N*sizeof(int)<<" B\n";
	
	// start general timer 
	const auto start = std::chrono::steady_clock::now();
	
	int hx[N] = {1,8,3,2,5,7,4,6};
	int *hy = 0;
	
	int *gx, *gy;
	
	cudaErrorCheck( hipMalloc((void**)&gx, N*sizeof(int)));
	cudaErrorCheck( hipMalloc((void**)&gy, sizeof(int)));
	
	 // end for malloc
	const auto mem_alloc_end = std::chrono::steady_clock::now();

	//Will be used to obtain a seed for the random number engine

	// copy the arrays from the CPU to the GPU
	hipMemcpy(gx, hx, N*sizeof(int), hipMemcpyHostToDevice);


	// end for initialization and memcpy
	const auto data_init_end = std::chrono::steady_clock::now();

	// Run kernel on 1 GiB of data on a single thread on the GPU
	ConvergentSumReductionKernel<<<1, 4>>>(gx, gy);

	cudaErrorCheck( hipPeekAtLastError() ); //debug the kernel output
	cudaErrorCheck( hipDeviceSynchronize() );

	const auto kernel_end = std::chrono::steady_clock::now();

	// copy the resulting array back to the GPU
	hipMemcpy(hy, gy, sizeof(int), hipMemcpyDeviceToHost);


	// Free memory
	hipFree(gx);
	hipFree(gy);

	std::cout << "Result: " << hy << "\n";

	//delete [] hx;
	delete [] hy;

	const auto free_mem = std::chrono::steady_clock::now();
	

	std::cout << "Memory allocation duration: " << std::chrono::duration_cast<std::chrono::microseconds>(mem_alloc_end-start).count()/1000<< " ms \n" \
		    << "Data initialization duration: " <<  std::chrono::duration_cast<std::chrono::microseconds>(data_init_end-mem_alloc_end).count()/1000 << " ms \n" \
		    << "Kernel duration: " << std::chrono::duration_cast<std::chrono::microseconds>(kernel_end-data_init_end).count() << " microseconds \n" \
		    << "Free memory duration: " << std::chrono::duration_cast<std::chrono::microseconds>(free_mem - kernel_end).count()/1000 << " ms \n";
		    
	hipDeviceReset();

	return 0;
}


