
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <random>

// Error checking for CUDA methods (not for the kernel though)
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void SimpleSumReductionKernel (int* input, int* output){
    unsigned int i = 2*threadIdx.x;
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2){
        if (threadIdx.x % stride == 0){
            input[i] += input[i + stride]; 
            //printf("Thread %d writes %d on location %d \t", threadIdx.x, input[i], i);
        }
        __syncthreads();
    }
      
    if(threadIdx.x == 0){
    	//for(int i=0; i<blockDim.x; i++){
    		//printf("%d \t", input[i]);
    	//}
        *output = input[0];
        printf("Output: %d \n", *output);
    }
}

int main(){


	int N = 8;
	std::cout<<"Size: "<< N*sizeof(int)<<" B\n";
	
	// start general timer 
	const auto start = std::chrono::steady_clock::now();
	
	// int hx[8] = {6,4,7,5,2,3,8,1};
	int hx[8] = {1,8,3,2,5,7,4,6};
	int *hy = 0;
	
	int *gx, *gy;
	
	cudaErrorCheck( hipMalloc((void**)&gx, N*sizeof(int)));
	cudaErrorCheck( hipMalloc((void**)&gy, sizeof(int)));
	
	 // end for malloc
	const auto mem_alloc_end = std::chrono::steady_clock::now();

	//Will be used to obtain a seed for the random number engine
	// std::random_device rdx;

	//Standard mersenne_twister_engine seeded with rd()
	// std::mt19937 genx(rdx()); 
	// std::uniform_int_distribution<> distrib(1, 100);

	// initialize hx and hy arrays on the host
	//for (int i = 0; i < N; i++) {
	//	hx[i] = i;
	//}
	
	
	//std::cout << "hx[0] = " << hx[0] << "\n";
	
	// copy the arrays from the CPU to the GPU
	hipMemcpy(gx, hx, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(hx, gx, sizeof(int), hipMemcpyDeviceToHost);

	// std::cout << "gx[0] = " << gx[0] << "\n";
	//std::cout << "hx[0] = " << hx[0] << "\n";

	// end for initialization and memcpy
	const auto data_init_end = std::chrono::steady_clock::now();

	// Run kernel on 1 GiB of data on a single thread on the GPU
	SimpleSumReductionKernel<<<1, 4>>>(gx, gy);

	cudaErrorCheck( hipPeekAtLastError() ); //debug the kernel output
	cudaErrorCheck( hipDeviceSynchronize() );

	const auto kernel_end = std::chrono::steady_clock::now();

	// copy the resulting array back to the GPU
	hipMemcpy(&hy, &gy, sizeof(int), hipMemcpyDeviceToHost);
	
	//std::cout << "Result: " << gy << "\n";
	//std::cout << "Result: " << &hy << "\n";

	// Free memory
	hipFree(gx);
	hipFree(gy);

	

	//delete [] hx;
	delete [] hy;

	const auto free_mem = std::chrono::steady_clock::now();
	

	std::cout << "Memory allocation duration: " << std::chrono::duration_cast<std::chrono::microseconds>(mem_alloc_end-start).count()/1000<< " ms \n" \
		    << "Data initialization duration: " <<  std::chrono::duration_cast<std::chrono::microseconds>(data_init_end-mem_alloc_end).count()/1000 << " ms \n" \
		    << "Kernel duration: " << std::chrono::duration_cast<std::chrono::microseconds>(kernel_end-data_init_end).count() << " microseconds \n" \
		    << "Free memory duration: " << std::chrono::duration_cast<std::chrono::microseconds>(free_mem - kernel_end).count()/1000 << " ms \n";

	return 0;
}
