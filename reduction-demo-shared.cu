
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>
#include <random>

#define N 8
#define BLOCK_DIM N/2

// Error checking for CUDA methods (not for the kernel though)
#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void SimpleSumReductionKernel (int* gx, int* gy){
    __shared__ int gx_s[BLOCK_DIM];
    unsigned int t = threadIdx.x;
    gx_s[t] = gx[t] + gx[t + BLOCK_DIM];
    for (unsigned int stride = blockDim.x/2; stride >= 1; stride /= 2){
        __syncthreads();
        if (threadIdx.x < stride){
            gx_s[t] += gx_s[t + stride];
            printf("Thread %d writes %d on location %d \t", threadIdx.x, gx_s[t], t);
        }
    }
    if(threadIdx.x == 0){
        *gy = gx_s[0];
    }
}

int main(){


	std::cout<<"Size: "<< N*sizeof(int)<<" B\n";
	
	// start general timer 
	const auto start = std::chrono::steady_clock::now();
	
	int hx[N] = {1,8,3,2,5,7,4,6};
	int *hy = 0;
	
	int *gx, *gy;
	
	cudaErrorCheck( hipMalloc((void**)&gx, N*sizeof(int)));
	cudaErrorCheck( hipMalloc((void**)&gy, sizeof(int)));
	
	 // end for malloc
	const auto mem_alloc_end = std::chrono::steady_clock::now();

	
	// copy the arrays from the CPU to the GPU
	hipMemcpy(gx, hx, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(hx, gx, sizeof(int), hipMemcpyDeviceToHost);

	// std::cout << "gx[0] = " << gx[0] << "\n";
	//std::cout << "hx[0] = " << hx[0] << "\n";

	// end for initialization and memcpy
	const auto data_init_end = std::chrono::steady_clock::now();

	// Run kernel on 1 GiB of data on a single thread on the GPU
	SimpleSumReductionKernel<<<1, BLOCK_DIM>>>(gx, gy);

	cudaErrorCheck( hipPeekAtLastError() ); //debug the kernel output
	cudaErrorCheck( hipDeviceSynchronize() );

	const auto kernel_end = std::chrono::steady_clock::now();

	// copy the resulting array back to the GPU
	hipMemcpy(hy, gy, sizeof(int), hipMemcpyDeviceToHost);
	
	// Free memory
	hipFree(gx);
	hipFree(gy);

	delete [] hy;

	const auto free_mem = std::chrono::steady_clock::now();
	

	std::cout << "Memory allocation duration: " << std::chrono::duration_cast<std::chrono::microseconds>(mem_alloc_end-start).count()/1000<< " ms \n" \
		    << "Data initialization duration: " <<  std::chrono::duration_cast<std::chrono::microseconds>(data_init_end-mem_alloc_end).count()/1000 << " ms \n" \
		    << "Kernel duration: " << std::chrono::duration_cast<std::chrono::microseconds>(kernel_end-data_init_end).count() << " microseconds \n" \
		    << "Free memory duration: " << std::chrono::duration_cast<std::chrono::microseconds>(free_mem - kernel_end).count()/1000 << " ms \n";
		    
	hipDeviceReset();

	return 0;
}


